#include "hip/hip_runtime.h"
#include <stdio.h>
#include <random>
#include "hip/hip_runtime.h"
#include ""

long long N = 1 << 30;
long long threads = 256;
long long blocks = (N + threads * 300 - 1) / (threads * 300);

__global__ void collatz(bool* x, long long N) {
    long long tid = blockDim.x * blockIdx.x + threadIdx.x;
    tid *= 300;
    if (tid + 299 < N && !x[tid+299]) {
        if (x[tid]) {
            bool b = x[tid];
            bool c = true;
            for (int k = tid + 1; k < tid + 300; k++) {
                bool a = x[k];
                x[k] = a ^ b ^ c;
                c = ((a ^ b) & c) | (a & b);
                b = a;
            }
            x[tid] = false;
        }
        else {
            for (int k = tid + 1; k < tid + 300; k++) {
                x[k - 1] = x[k];
            }
            x[tid + 299] = false;
        }
    }
}

void prepare_input_data(bool A[], int n) {
    std::default_random_engine gen(20240312);
    std::uniform_real_distribution<float> fran(0.0, 1.0);
    for (int k = 0; k < n; k++) {
        A[k] = (k % 300 <= 111) ? (fran(gen) >= 0.5) : false;
    }
}

int main(int argc, char* argv[]) {
    fprintf(stdout, "blocks : %lld\nthreads : %lld\n\n", blocks, threads);
    bool* h_A = new bool[N];
    prepare_input_data(h_A, N);
    bool* d_A;
    hipMalloc((void**)&d_A, N * sizeof(bool));
    hipMemcpy(d_A, h_A, N * sizeof(bool), hipMemcpyHostToDevice);

    for (int i = 0; i < 2000; i++) {
        collatz << <blocks, threads >> > (d_A, N);
        hipDeviceSynchronize();
    }
    hipMemcpy(h_A, d_A, N * sizeof(bool), hipMemcpyDeviceToHost);

    for (int i = 0; i < N / 300; i++) {
        for (int k = i * 300 + 299; k >= i * 300 + 64; k--) {
            if (h_A[k]) {
                for (int l = i * 300 + 299; l >= i * 300; l--) {
                    printf("%d", h_A[l] ? 1 : 0);
                }
                printf("\n");
                break;
            }
        }
    }
    hipFree(d_A);
    delete[] h_A;
    return 0;
}
